
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <sys/time.h>

// Returns the current system time in microseconds 
long long get_time()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (tv.tv_sec * 1000000) + tv.tv_usec;

}

using namespace std;

#define BLOCK_SIZE 16
#define BLOCK_SIZE_C BLOCK_SIZE
#define BLOCK_SIZE_R BLOCK_SIZE

#define STR_SIZE	256

/* maximum power density possible (say 300W for a 10mm x 10mm chip)	*/
#define MAX_PD	(3.0e6)
/* required precision in degrees	*/
#define PRECISION	0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor	*/
#define FACTOR_CHIP	0.5
#define OPEN
//#define NUM_THREAD 4

typedef float FLOAT;

void kernel_ifs(FLOAT *result, FLOAT *temp, FLOAT *power, int col, int row, FLOAT Cap_1, FLOAT Rx_1, 
				FLOAT Ry_1, FLOAT Rz_1, FLOAT amb_temp);

/* chip parameters	*/
const FLOAT t_chip = 0.0005;
const FLOAT chip_height = 0.016;
const FLOAT chip_width = 0.016;

/* ambient temperature, assuming no package at all	*/
const FLOAT amb_temp = 80.0;

int num_omp_threads;


__constant__ FLOAT amb_temp_dev;

#define THREADS_PER_BLOCK 512

__global__ void kernel ( FLOAT *Ry_1_dev, FLOAT *Rx_1_dev, FLOAT *Rz_1_dev, FLOAT* Cap_1_dev, int* size_dev,
        FLOAT *result_dev, FLOAT *temp_dev, FLOAT *power_dev, FLOAT* col_minus_1_dev, FLOAT* col_plus_1_dev) {

    unsigned int column = blockIdx.x*blockDim.x + threadIdx.x+BLOCK_SIZE;
    unsigned int row = blockIdx.y+BLOCK_SIZE;
    
    
    int size = *size_dev;
	
	
	if (column == BLOCK_SIZE)
	{
		result_dev[row*size+column] =temp_dev[row*size+column]+ 
			( (*Cap_1_dev) * (power_dev[row*size+column] + 
			(temp_dev[(row+1)*size+column] + temp_dev[(row-1)*size+column] - 2.f*temp_dev[row*size+column]) * (*Ry_1_dev) + 
			(temp_dev[row*size+column+1] + col_minus_1_dev[row] - 2.f*temp_dev[row*size+column]) * (*Rx_1_dev) + 
			(amb_temp_dev - temp_dev[row*size+column]) * (*Rz_1_dev)));
	}
	else if (column == size - BLOCK_SIZE - 1)
	{
		result_dev[row*size+column] =temp_dev[row*size+column]+ 
			( (*Cap_1_dev) * (power_dev[row*size+column] + 
			(temp_dev[(row+1)*size+column] + temp_dev[(row-1)*size+column] - 2.f*temp_dev[row*size+column]) * (*Ry_1_dev) + 
			(col_plus_1_dev[row] + temp_dev[row*size+column-1] - 2.f*temp_dev[row*size+column]) * (*Rx_1_dev) + 
			(amb_temp_dev - temp_dev[row*size+column]) * (*Rz_1_dev)));
	}
	else{ //if (row < size - 15  && row > 15) {
	result_dev[row*size+column] =temp_dev[row*size+column]+ 
		 ( (*Cap_1_dev) * (power_dev[row*size+column] + 
		(temp_dev[(row+1)*size+column] + temp_dev[(row-1)*size+column] - 2.f*temp_dev[row*size+column]) * (*Ry_1_dev) + 
		(temp_dev[row*size+column+1] + temp_dev[row*size+column-1] - 2.f*temp_dev[row*size+column]) * (*Rx_1_dev) + 
		(amb_temp_dev - temp_dev[row*size+column]) * (*Rz_1_dev)));
	}
    

}

/* Transient solver driver routine: simply converts the heat 
 * transfer differential equations to difference equations 
 * and solves the difference equations by iterating
 */
void compute_tran_temp(FLOAT *result, int num_iterations, FLOAT *temp, FLOAT *power, int row, int col) 
{
	#ifdef VERBOSE
	int i = 0;
	#endif
    long soma = 0;

	FLOAT grid_height = chip_height / row;
	FLOAT grid_width = chip_width / col;

	FLOAT Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * grid_width * grid_height;
	FLOAT Rx = grid_width / (2.0 * K_SI * t_chip * grid_height);
	FLOAT Ry = grid_height / (2.0 * K_SI * t_chip * grid_width);
	FLOAT Rz = t_chip / (K_SI * grid_height * grid_width);

	FLOAT max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
    FLOAT step = PRECISION / max_slope / 1000.0;

    FLOAT Rx_1=1.f/Rx;
    FLOAT Ry_1=1.f/Ry;
    FLOAT Rz_1=1.f/Rz;
    FLOAT Cap_1 = step/Cap;
	FLOAT *col_minus_1, *col_plus_1;
	#ifdef VERBOSE
	fprintf(stdout, "total iterations: %d s\tstep size: %g s\n", num_iterations, step);
	fprintf(stdout, "Rx: %g\tRy: %g\tRz: %g\tCap: %g\n", Rx, Ry, Rz, Cap);
	#endif
	
	//cudaMallocHost( (FLOAT **) &col_minus_1 , col* sizeof(FLOAT) );
	//cudaMallocHost( (FLOAT **) &col_plus_1 , col* sizeof(FLOAT) );
	col_minus_1=(FLOAT *) calloc (col, sizeof(FLOAT));
    col_plus_1=(FLOAT *) calloc (col, sizeof(FLOAT));

    

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // alocar memoria no gpu
    FLOAT *result_dev = NULL;
    err = hipMalloc((void **)&result_dev, (size_t)(sizeof(FLOAT)*col*row));
    FLOAT *power_dev = NULL;
    err = hipMalloc((void **)&power_dev, (size_t)(sizeof(FLOAT)*row*col));
    FLOAT *temp_dev = NULL;
    err = hipMalloc((void **)&temp_dev, (size_t)(sizeof(FLOAT)*row*col));
    FLOAT *col_minus_1_dev = NULL;
    err = hipMalloc((void **)&col_minus_1_dev, (size_t)(sizeof(FLOAT)*col));
    FLOAT *col_plus_1_dev = NULL;
    err = hipMalloc((void **)&col_plus_1_dev, (size_t)(sizeof(FLOAT)*col));
    FLOAT *Ry_1_dev = NULL;
    err = hipMalloc((void **)&Ry_1_dev, (size_t)sizeof(FLOAT));
    FLOAT *Rx_1_dev = NULL;
    err = hipMalloc((void **)&Rx_1_dev, (size_t)sizeof(FLOAT));
    FLOAT *Rz_1_dev = NULL;
    err = hipMalloc((void **)&Rz_1_dev, (size_t)sizeof(FLOAT));
    FLOAT *Cap_1_dev = NULL;
    err = hipMalloc((void **)&Cap_1_dev, (size_t)sizeof(FLOAT));
    int *size_dev = NULL;
    err = hipMalloc((void **)&size_dev, (size_t)sizeof(int));
	
    //transferir para o gpu
    err = hipMemcpy(temp_dev, temp, (size_t)(sizeof(FLOAT)*col*row), hipMemcpyHostToDevice);
    err = hipMemcpy(power_dev, power, (size_t)(sizeof(FLOAT)*col*row), hipMemcpyHostToDevice);
    
    err = hipMemcpy(Ry_1_dev, &Ry_1, (size_t)sizeof(FLOAT), hipMemcpyHostToDevice);
    err = hipMemcpy(Rx_1_dev, &Rx_1, (size_t)sizeof(FLOAT), hipMemcpyHostToDevice);
    err = hipMemcpy(Rz_1_dev, &Rz_1, (size_t)sizeof(FLOAT), hipMemcpyHostToDevice);
    err = hipMemcpy(Cap_1_dev, &Cap_1, (size_t)sizeof(FLOAT), hipMemcpyHostToDevice);
    err = hipMemcpy(size_dev, &col, (size_t)sizeof(int), hipMemcpyHostToDevice);
    //copy amb_temp to device
    hipMemcpyToSymbol(HIP_SYMBOL(amb_temp_dev), &amb_temp, (size_t)sizeof(FLOAT));
    

    dim3 blockDist(THREADS_PER_BLOCK,1,1);
    dim3 gridDist((row-(2*BLOCK_SIZE)+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, col-2*BLOCK_SIZE, 1);


    FLOAT* r = result;
    FLOAT* t = temp;
    FLOAT* tmp;
    for (int i = 0; i < num_iterations ; i++)
    {
        #ifdef VERBOSE
        fprintf(stdout, "iteration %d\n", i++);
        #endif
        result = r;

        
        if (i!=0)
		{
			err = hipMemcpyAsync((temp_dev+(BLOCK_SIZE-1)*col), (temp+(BLOCK_SIZE-1)*col), (size_t)(sizeof(FLOAT)*col), hipMemcpyHostToDevice);
			err = hipMemcpyAsync((temp_dev+(row-BLOCK_SIZE)*col), (temp+(row-BLOCK_SIZE)*col), (size_t)(sizeof(FLOAT)*col), hipMemcpyHostToDevice);
			

			for (int j = 0; j < row; j++) {
				col_minus_1[j] = *(temp + j*row+BLOCK_SIZE-1);
				col_plus_1[j] = *(temp + j*row + col-BLOCK_SIZE);
			}

			err = hipMemcpyAsync(col_minus_1_dev, col_minus_1, (size_t)(sizeof(FLOAT)*row), hipMemcpyHostToDevice);
			err = hipMemcpyAsync(col_plus_1_dev, col_plus_1, (size_t)(sizeof(FLOAT)*row), hipMemcpyHostToDevice);
		}

 
        kernel<<<gridDist, blockDist>>> (Ry_1_dev, Rx_1_dev, Rz_1_dev, Cap_1_dev, size_dev,
                result_dev, temp_dev, power_dev, col_minus_1_dev, col_plus_1_dev);

        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));                
            exit(EXIT_FAILURE);
        }

        if (i == num_iterations-1) 
            err = hipMemcpy(result, result_dev, (size_t)(sizeof(FLOAT)*col*row), hipMemcpyDeviceToHost);
        
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy vector result from device to host (error code %s)!\n", hipGetErrorString(err));      
            exit(EXIT_FAILURE);
        }

        kernel_ifs(result, temp, power, col, row, Cap_1, Rx_1, Ry_1, Rz_1, amb_temp);
        
        tmp = temp;
        temp = result;
        r = tmp;
        
        FLOAT* tmp_dev = temp_dev;
        temp_dev = result_dev;
        result_dev = tmp_dev;
    }	

    hipFree(result_dev);
    hipFree(temp_dev);
    hipFree(power_dev);
    hipFree(Cap_1_dev);
    hipFree(Ry_1_dev);
    hipFree(Rx_1_dev);
    hipFree(Rz_1_dev);
    hipFree(size_dev);
	hipHostFree(col_minus_1);
    hipHostFree(col_plus_1);

	#ifdef VERBOSE
	fprintf(stdout, "iteration %d\n", i++);
	#endif
}

void fatal(const char *s)
{
	fprintf(stderr, "error: %s\n", s);
	exit(1);
}

void writeoutput(FLOAT *vect, int grid_rows, int grid_cols, char *file) {

    int i,j, index=0;
    FILE *fp;
    char str[STR_SIZE];

    if( (fp = fopen(file, "w" )) == 0 )
        printf( "The file was not opened\n" );


    for (i=0; i < grid_rows; i++) 
        for (j=0; j < grid_cols; j++)
        {

            sprintf(str, "%d\t%g\n", index, vect[i*grid_cols+j]);
            fputs(str,fp);
            index++;
        }

    fclose(fp);	
}

void read_input(FLOAT *vect, int grid_rows, int grid_cols, char *file)
{
  	int i, index;
	FILE *fp;
	char str[STR_SIZE];
	FLOAT val;

	fp = fopen (file, "r");
	if (!fp)
		fatal("file could not be opened for reading");

	for (i=0; i < grid_rows * grid_cols; i++) {
		fgets(str, STR_SIZE, fp);
		if (feof(fp))
			fatal("not enough lines in file");
		if ((sscanf(str, "%f", &val) != 1) )
			fatal("invalid file format");
		vect[i] = val;
	}

	fclose(fp);	
}

void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <grid_rows> <grid_cols> <sim_time> <no. of threads><temp_file> <power_file>\n", argv[0]);
	fprintf(stderr, "\t<grid_rows>  - number of rows in the grid (positive integer)\n");
	fprintf(stderr, "\t<grid_cols>  - number of columns in the grid (positive integer)\n");
	fprintf(stderr, "\t<sim_time>   - number of iterations\n");
	fprintf(stderr, "\t<no. of threads>   - number of threads\n");
	fprintf(stderr, "\t<temp_file>  - name of the file containing the initial temperature values of each cell\n");
	fprintf(stderr, "\t<power_file> - name of the file containing the dissipated power values of each cell\n");
        fprintf(stderr, "\t<output_file> - name of the output file\n");
	exit(1);
}

int main(int argc, char **argv)
{
	int grid_rows, grid_cols, sim_time, i;
	FLOAT *temp, *power, *result;
	char *tfile, *pfile, *ofile;
	
	/* check validity of inputs	*/
	if (argc != 8)
		usage(argc, argv);
	if ((grid_rows = atoi(argv[1])) <= 0 ||
		(grid_cols = atoi(argv[2])) <= 0 ||
		(sim_time = atoi(argv[3])) <= 0 || 
		(num_omp_threads = atoi(argv[4])) <= 0
		)
		usage(argc, argv);

	/* allocate memory for the temperature and power arrays	*/
	//cudaMallocHost( (FLOAT **) &temp , grid_rows *grid_cols* sizeof(FLOAT) );
	//cudaMallocHost( (FLOAT **) &power , grid_rows *grid_cols* sizeof(FLOAT) );
	//cudaMallocHost( (FLOAT **) &result , grid_rows *grid_cols* sizeof(FLOAT) );
	temp=(FLOAT *) calloc (grid_rows *grid_cols, sizeof(FLOAT));
    power=(FLOAT *) calloc (grid_rows *grid_cols, sizeof(FLOAT));
	result=(FLOAT *) calloc (grid_rows *grid_cols, sizeof(FLOAT));

	if(!temp || !power)
		fatal("unable to allocate memory");

	/* read initial temperatures and input power	*/
	tfile = argv[5];
	pfile = argv[6];
    ofile = argv[7];

	read_input(temp, grid_rows, grid_cols, tfile);
	read_input(power, grid_rows, grid_cols, pfile);

	printf("Start computing the transient temperature\n");
	
    long long start_time = get_time();

    compute_tran_temp(result,sim_time, temp, power, grid_rows, grid_cols);

    long long end_time = get_time();

    printf("Ending simulation\n");
    printf("Total time: %.3f seconds\n", ((float) (end_time - start_time)) / (1000*1000));

    writeoutput((1&sim_time) ? result : temp, grid_rows, grid_cols, ofile);

	/* output results	*/
#ifdef VERBOSE
	fprintf(stdout, "Final Temperatures:\n");
#endif

#ifdef OUTPUT
	for(i=0; i < grid_rows * grid_cols; i++)
	fprintf(stdout, "%d\t%g\n", i, temp[i]);
#endif
	/* cleanup	*/
	hipHostFree(temp);
	hipHostFree(power);

	return 0;
}



void kernel_ifs(FLOAT *result, FLOAT *temp, FLOAT *power, int col, int row, FLOAT Cap_1, FLOAT Rx_1, 
				FLOAT Ry_1, FLOAT Rz_1, FLOAT amb_temp)
{
    FLOAT delta;
    int r,c;
    int chunk;
    int num_chunk = row*col / (BLOCK_SIZE_R * BLOCK_SIZE_C);
    int chunks_in_row = col/BLOCK_SIZE_C;
    int chunks_in_col = row/BLOCK_SIZE_R;
	
	for ( chunk = 0; chunk < num_chunk; ++chunk )
	{
		int r_start = BLOCK_SIZE_R*(chunk/chunks_in_col);
		int c_start = BLOCK_SIZE_C*(chunk%chunks_in_row); 
		int r_end = r_start + BLOCK_SIZE_R > row ? row : r_start + BLOCK_SIZE_R;
		int c_end = c_start + BLOCK_SIZE_C > col ? col : c_start + BLOCK_SIZE_C;
	   
	   
		if ( r_start == 0 || c_start == 0 || r_end == row || c_end == col )
		{	
			for (  r = r_start; r < r_start + BLOCK_SIZE_R; ++r ) 
			{
                for ( c = c_start; c < c_start + BLOCK_SIZE_C; ++c ) {
                    /* Corner 1 */
                    if ( (r == 0) && (c == 0) ) {
                        delta = (Cap_1) * (power[0] +
                            (temp[1] - temp[0]) * Rx_1 +
                            (temp[col] - temp[0]) * Ry_1 +
                            (amb_temp - temp[0]) * Rz_1);
                    }	/* Corner 2 */
                    else if ((r == 0) && (c == col-1)) {
                        delta = (Cap_1) * (power[c] +
                            (temp[c-1] - temp[c]) * Rx_1 +
                            (temp[c+col] - temp[c]) * Ry_1 +
                        (   amb_temp - temp[c]) * Rz_1);
                    }	/* Corner 3 */
                    else if ((r == row-1) && (c == col-1)) {
                        delta = (Cap_1) * (power[r*col+c] + 
                            (temp[r*col+c-1] - temp[r*col+c]) * Rx_1 + 
                            (temp[(r-1)*col+c] - temp[r*col+c]) * Ry_1 + 
                        (   amb_temp - temp[r*col+c]) * Rz_1);					
                    }	/* Corner 4	*/
                    else if ((r == row-1) && (c == 0)) {
                        delta = (Cap_1) * (power[r*col] + 
                            (temp[r*col+1] - temp[r*col]) * Rx_1 + 
                            (temp[(r-1)*col] - temp[r*col]) * Ry_1 + 
                            (amb_temp - temp[r*col]) * Rz_1);
                    }	/* Edge 1 */
                    else if (r == 0) {
                        delta = (Cap_1) * (power[c] + 
                            (temp[c+1] + temp[c-1] - 2.0*temp[c]) * Rx_1 + 
                            (temp[col+c] - temp[c]) * Ry_1 + 
                            (amb_temp - temp[c]) * Rz_1);
                    }	/* Edge 2 */
                    else if (c == col-1) {
                        delta = (Cap_1) * (power[r*col+c] + 
                            (temp[(r+1)*col+c] + temp[(r-1)*col+c] - 2.0*temp[r*col+c]) * Ry_1 + 
                            (temp[r*col+c-1] - temp[r*col+c]) * Rx_1 + 
                            (amb_temp - temp[r*col+c]) * Rz_1);
                    }	/* Edge 3 */
                    else if (r == row-1) {
                        delta = (Cap_1) * (power[r*col+c] + 
                            (temp[r*col+c+1] + temp[r*col+c-1] - 2.0*temp[r*col+c]) * Rx_1 + 
                            (temp[(r-1)*col+c] - temp[r*col+c]) * Ry_1 + 
                            (amb_temp - temp[r*col+c]) * Rz_1);
                    }	/* Edge 4 */
                    else if (c == 0) {
                        delta = (Cap_1) * (power[r*col] + 
                            (temp[(r+1)*col] + temp[(r-1)*col] - 2.0*temp[r*col]) * Ry_1 + 
                            (temp[r*col+1] - temp[r*col]) * Rx_1 + 
                            (amb_temp - temp[r*col]) * Rz_1);
                    }
                    result[r*col+c] =temp[r*col+c]+ delta;
                }

			}
		}
	}
    	
}	

/* vim: set ts=4 sw=4  sts=4 et si ai: */
